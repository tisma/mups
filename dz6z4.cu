#include "hip/hip_runtime.h"
/******************************
 * Tisma Miroslav 2006/0395
 * Multiprocesorski sistemi
 * domaci zadatak 6 - 4. zadatak
 *******************************/

/**
 * 4. Sastaviti program koji menja znak svim elementima niza celih brojeva. Po zavrsenoj obradi niza, treba 
 * ispisati izmenjeni niz, ukupan broj pozitivnih i ukupan broj negativnih elemenata rezultujuceg niza. 
 */

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define NUM_OF_GPU_THREADS 256

__global__ void changeSignsInArray(int *array1, int *positive, int *negative, int n) {
	int i, counter = 0;
	int idx = threadIdx.x;

	__shared__ int numbers[NUM_OF_GPU_THREADS];

	int slice = n / NUM_OF_GPU_THREADS;
	int start = idx * slice;
	if (idx == NUM_OF_GPU_THREADS - 1)
		slice += n % NUM_OF_GPU_THREADS;
	int end = start + slice;

	for (i = start; i < end; i++) {
		array1[i] = -array1[i];
		if (array1[i] > 0)
			counter++;
	}

	numbers[idx] = counter;
	__syncthreads();

	int half = NUM_OF_GPU_THREADS;
	do {
		__syncthreads();
		half >>= 1;
		if (idx < half)
			numbers[idx] += numbers[idx + half];
	} while(half != 1);

	if (idx == 0) {
		*positive = numbers[0];
		*negative = n - *positive;
	}
}

int main(int argc, char *argv[]) {
	
	int i, n, h_positives, h_negatives;
	int *h_array;
	int *d_array, *d_positives, *d_negatives;
	FILE *out;

	printf("Nizovi ce biti upisani u fajl dz6_4_izl1.in\n");
	printf("Unesite velicinu niza:\n");
	scanf("%d", &n);

	h_array = (int*)malloc(n*sizeof(int));

	out = fopen("dz6_4_izl1.in", "w");
	if (out == NULL) {
		printf("Greska pri otvaranju fajla!");
		exit(EXIT_FAILURE);
	}
	fprintf(out, "ORIGINALNI NIZ\n");

	srand(time(0));
	for (i = 0; i < n; i++) {
		h_array[i] = -100 + rand() % 200;
		fprintf(out, "%3d ", h_array[i]);
	}
	fprintf(out, "\n");

	hipMalloc((void**)&d_array, n*sizeof(int));
	hipMalloc((void**)&d_positives, sizeof(int));
	hipMalloc((void**)&d_negatives, sizeof(int));

	hipMemcpy(d_array, h_array, n*sizeof(int), hipMemcpyHostToDevice);

	changeSignsInArray<<<1, NUM_OF_GPU_THREADS>>>(d_array, d_positives, d_negatives, n);

	hipDeviceSynchronize();

	hipMemcpy(h_array, d_array, n*sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&h_positives, d_positives, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&h_negatives, d_negatives, sizeof(int), hipMemcpyDeviceToHost);

	fprintf(out, "IZMENJENI NIZ:\n");
	for (i = 0; i < n; i++)
		fprintf(out, "%3.2d ", h_array[i]);

	fprintf(out, "\n");
	fprintf(out, "Broj pozitivnih elemenata u nizu je: %d\n", h_positives);
	fprintf(out, "Broj negativnih elemenata u nizu je: %d\n", h_negatives);

	hipFree(d_array);
	hipFree(d_positives);
	hipFree(d_negatives);
	free(h_array);
	fclose(out);

	return EXIT_SUCCESS;
}