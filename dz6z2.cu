#include "hip/hip_runtime.h"
/******************************
 * Tisma Miroslav 2006/0395
 * Multiprocesorski sistemi
 * domaci zadatak 6 - 2. zadatak
 *******************************/

/**
 * 2. Sastaviti program koji pronalazi najmanji i najveci element dvodimenzionalne matrice.
 */

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define NUM_OF_GPU_THREADS 256

__global__ void findMinMaxInMatrix(int *matrix, int *min, int *max, int size) {

	int i;
	int idx = threadIdx.x;

	__shared__ int maxs[NUM_OF_GPU_THREADS];
	__shared__ int mins[NUM_OF_GPU_THREADS];

	int slice = size / NUM_OF_GPU_THREADS;
	int start = idx * slice;
	if (idx == NUM_OF_GPU_THREADS - 1)
		slice += size % NUM_OF_GPU_THREADS;
	int end = start + slice;

	int local_min = matrix[start];
	int local_max = matrix[start];

	for (i = start; i < end; i++) {
		if (matrix[i] < local_min)
			local_min = matrix[i];
		else if (matrix[i] > local_max)
			local_max = matrix[i];
	}

	mins[idx] = local_min;
	maxs[idx] = local_max;
	__syncthreads();
	
	int half = NUM_OF_GPU_THREADS;
	do {
		__syncthreads();
		half >>= 1;
		if (idx < half) {
			if (mins[idx] < mins[idx + half])
				mins[idx] = mins[idx];
			else
				mins[idx] = mins[idx + half];
			if (maxs[idx] > maxs[idx + half])
				maxs[idx] = maxs[idx];
			else
				maxs[idx] = maxs[idx + half];
		}
	} while(half != 1);

	if (idx == 0) {
		*min = mins[0];
		*max = maxs[0];
	}
}

int main(int argc, char *argv[]) {

	int i, m, n;
	int min, max;
	int *h_matrix, *d_matrix, *d_min, *d_max;

	printf("Matrica ce biti ispisana na standardnom izlazu\n");
	printf("Unesite broj vrsta matrice:\n");
	scanf("%d", &m);
	printf("Unesite broj kolona matrice:\n");
	scanf("%d", &n);

	h_matrix = (int*)malloc(m*n*sizeof(int));
	srand(time(0));
	for (i = 0; i < m*n; i++) {
		if (i % m == 0)
			printf("\n");
		h_matrix[i] = -10000 + rand() % 20000;
		printf("%5d ", h_matrix[i]);
	}
	printf("\n");

	hipMalloc((void**)&d_matrix, m*n*sizeof(int));
	hipMalloc((void**)&d_min, sizeof(int));
	hipMalloc((void**)&d_max, sizeof(int));

	hipMemcpy(d_matrix, h_matrix, m*n*sizeof(int), hipMemcpyHostToDevice);

	findMinMaxInMatrix<<< 1, NUM_OF_GPU_THREADS >>>(d_matrix, d_min, d_max, m*n);

	hipDeviceSynchronize();

	hipMemcpy(&min, d_min, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(&max, d_max, sizeof(int), hipMemcpyDeviceToHost);

	printf("\nNajmanji element u matrici je: %d\n", min);
	printf("Najveci element u matrici je: %d\n", max);

	hipFree(d_min);
	hipFree(d_max);
	hipFree(d_matrix);
	free(h_matrix);

	return EXIT_SUCCESS;
}


