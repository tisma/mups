#include "hip/hip_runtime.h"
/******************************
 * Tisma Miroslav 2006/0395
 * Multiprocesorski sistemi
 * domaci zadatak 6 - 1. zadatak
 *******************************/

/**
 * 1. Sastaviti program koji kvadrira elemente dvodimenzionalne matrice.
 */

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define NUM_OF_GPU_THREADS 256

__global__ void matrixSquare(int *matrix) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	matrix[idx] = matrix[idx] * matrix[idx];
}

int main(int argc, char *argv[]) {
	int i, j, m, n;
	int *h_matrix, *d_matrix;
	FILE *out;

	printf("Matrice ce biti upisane u fajl dz6_1_izl1.in\n");
	printf("Unesite broj vrsta matrice:\n");
	scanf("%d", &m);
	printf("Unesite broj kolona matrice:\n");
	scanf("%d", &n);

	out = fopen("dz6_1_izl1.in", "w");
	if (out == NULL) {
		printf("Greska pri otvaranju fajla!");
		exit(EXIT_FAILURE);
	}

	fprintf(out, "ORIGINALNA MATRICA:\n");

	h_matrix = (int*) malloc(m*n * sizeof(int));
	for (i = 0; i < m*n; i++) {
		if (i % m == 0)
			fprintf(out, "\n");
		h_matrix[i] = -100 + rand() % 200;
		fprintf(out, "%4d ", h_matrix[i]);
	}

	int dimGrid(m*n / NUM_OF_GPU_THREADS + 1);
	int dimBlock(NUM_OF_GPU_THREADS);
	int size = (m*n * sizeof(int) / (dimGrid * dimBlock) + 1) * (dimGrid * dimBlock);

	hipMalloc((void**)&d_matrix, size);
	hipMemcpy(d_matrix, h_matrix, m*n * sizeof(int), hipMemcpyHostToDevice);
	matrixSquare<<<dimGrid, dimBlock>>>(d_matrix);
	
	hipDeviceSynchronize();

	hipMemcpy(h_matrix, d_matrix, m*n * sizeof(int), hipMemcpyDeviceToHost);

	fprintf(out, "\n\nKVADRIRANA MATRICA\n\n");
	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++)
			fprintf(out, "%5d ", h_matrix[i * m + j]);
		fprintf(out, "\n");
	}

	fclose(out);
	hipFree(d_matrix);

	return EXIT_SUCCESS;
}