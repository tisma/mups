#include "hip/hip_runtime.h"
/******************************
 * Tisma Miroslav 2006/0395
 * Multiprocesorski sistemi
 * domaci zadatak 6 - 5. zadatak
 *******************************/

/**
 * 5. Sastaviti program koji odredjuje vrednost broja PI na neki od dva ispod opisana nacina.
 */

#include "hip/hip_runtime.h"
#include ""
#include "hip/device_functions.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>

#define NUM_OF_GPU_THREADS 256

__global__ void calculatePi(int *array1, int n, float *pi) {
	int i;
	float sum = 0.0;
	int idx = threadIdx.x;

	__shared__ float numbers[NUM_OF_GPU_THREADS];

	int slice = n / NUM_OF_GPU_THREADS;
	int start = idx * slice;
	if (idx == NUM_OF_GPU_THREADS - 1)
		slice += n % NUM_OF_GPU_THREADS;
	int end = start + slice;

	for (i = start; i < end; i++) {
		if (i == 0) continue;	
		sum += (float)1.0 / (float)(i*i);
	}

	numbers[idx] = sum;
	__syncthreads();

	int half = NUM_OF_GPU_THREADS;
	do {
		__syncthreads();
		half >>= 1;
		if (idx < half)
			numbers[idx] += numbers[idx + half];
	} while(half != 1);

	if (idx == 0) {
		*pi = sqrt(6.0 * numbers[0]);
	}

}

int main(int argc, char *argv[]) {
	int i, n;
	int *h_array;
	int *d_array;
	float h_pi;
	float *d_pi;

	printf("Unesite broj tacaka za priblizno racunanje broja PI:\n");
	scanf("%d", &n);

	h_array = (int*)malloc(n*sizeof(int));
	
	srand(time(0));
	for (i = 0; i < n; i++)
		h_array[i] = rand();
		
	hipMalloc((void**)&d_array, n*sizeof(int));
	hipMalloc((void**)&d_pi, sizeof(float));

	hipMemcpy(d_array, h_array, n*sizeof(int), hipMemcpyHostToDevice);

	calculatePi<<<1, NUM_OF_GPU_THREADS>>>(d_array, n, d_pi);

	hipDeviceSynchronize();

	hipMemcpy(&h_pi, d_pi, sizeof(float), hipMemcpyDeviceToHost);

	printf("Priblizna vrednost broja PI je: %lf\n", h_pi);

	hipFree(d_array);
	hipFree(d_pi);
	free(h_array);

	return EXIT_SUCCESS;
}

