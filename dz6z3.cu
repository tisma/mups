#include "hip/hip_runtime.h"
/******************************
 * Tisma Miroslav 2006/0395
 * Multiprocesorski sistemi
 * domaci zadatak 6 - 3. zadatak
 *******************************/
/**
 * 3. Sastaviti program koji racuna skalarni proizvod dva niza
 */

#include "hip/hip_runtime.h"
#include "hip/device_functions.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#define NUM_OF_GPU_THREADS 256

__global__ void dotProduct(int *array1, int *array2, int n, int *result) {

	int i, sum = 0;
	int idx = threadIdx.x;

	__shared__ int dotSum[NUM_OF_GPU_THREADS];

	int slice = n / NUM_OF_GPU_THREADS;
	int start = idx * slice;
	if (idx == NUM_OF_GPU_THREADS - 1)
		slice += n % NUM_OF_GPU_THREADS;
	int end = start + slice;


	for (i = start; i < end; i++) {
		array1[i] = array1[i] * array2[i];
		sum += array1[i];
	}

	dotSum[idx] = sum;
	__syncthreads();

	int half = NUM_OF_GPU_THREADS;
	do {
		__syncthreads();
		half >>= 1;
		if (idx < half)
			dotSum[idx] += dotSum[idx + half];
	} while(half != 1);

	if (idx == 0)
		*result = dotSum[0];
}

int main(int argc, char *argv[]) {
	int i, n;
	int *h_array1, *h_array2;
	int *d_array1, *d_array2;
	int h_result;
	int *d_result;

	printf("Nizovi ce biti ispisani na standardnom izlazu\n");
	printf("Unesite velicinu nizova:\n");
	scanf("%d", &n);

	h_array1 = (int*)malloc(n*sizeof(int));
	h_array2 = (int*)malloc(n*sizeof(int));

	printf("\n");
	srand(time(0));
	for (i = 0; i < n; i++) {
		h_array1[i] = -100 + rand() % 200;
		printf("%2d ", h_array1[i]);
	}
	printf("\n");
	for (i = 0; i < n; i++) {
		h_array2[i] = -100 + rand() % 200;
		printf("%2d ", h_array2[i]);
	}
	printf("\n");

	hipMalloc((void**)&d_array1, n*sizeof(int));
	hipMalloc((void**)&d_array2, n*sizeof(int));
	hipMalloc((void**)&d_result, sizeof(int));

	hipMemcpy(d_array1, h_array1, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_array2, h_array2, n*sizeof(int), hipMemcpyHostToDevice);
	
	dotProduct<<<1, NUM_OF_GPU_THREADS>>>(d_array1, d_array2, n, d_result);

	hipDeviceSynchronize();

	hipMemcpy(&h_result, d_result, sizeof(int), hipMemcpyDeviceToHost);

	printf("\nSkalarni proizvod nizova je: %d\n", h_result);

	hipFree(d_array1);
	hipFree(d_array2);
	hipFree(d_result);
	free(h_array1);
	free(h_array2);

	return EXIT_SUCCESS;
}
